#include "hip/hip_runtime.h"
/*
   Copyright (C) 2011  Abhinav Jauhri (abhinav.jauhri@gmail.com), Carnegie Mellon University - Silicon Valley 

   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#define TILE_WIDTH 2

namespace cuda
{
	__device__ float* get_sub_matrix(float *sq_matrix, int sq_dimension, int row, int col){
		float *sub_matrix;
		sub_matrix = &sq_matrix[sq_dimension*row*BLOCK_SIZE + BLOCK_SIZE*col];
		return sub_matrix;
	}
	void padding_reverse(float *sq_matrix, float *new_matrix, unsigned int sq_dimension, unsigned int new_dimension) {
		for(int i=0; i<sq_dimension; i++) {
			for(int j=0; j<sq_dimension; j++) {
				sq_matrix[i*sq_dimension + j] = new_matrix[i*new_dimension + j];
			}
		}
	}
	void padding(float *sq_matrix, float *new_matrix, unsigned int sq_dimension, unsigned int new_dimension) {
		for(int i=0; i<new_dimension; i++) {
			for(int j=0; j<new_dimension; j++) {
				if(i<sq_dimension && j<sq_dimension) {
					new_matrix[i*new_dimension + j] = sq_matrix[i*sq_dimension + j];
				}
				else {
					new_matrix[i*new_dimension + j] = 0;
				}
			}
		}
	}
	__global__ 
		void 
		small_matrix_mul_kernel(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension)
		{
			int row = blockIdx.y*blockDim.y+threadIdx.y;
			int col = blockIdx.x*blockDim.x+threadIdx.x;

			float sum = 0.0f;

			for(int k = 0; k < sq_dimension; k++)
			{
				sum += sq_matrix_1[row*sq_dimension + k] * sq_matrix_2[k*sq_dimension + col];
			}
			sq_matrix_result[row*sq_dimension + col] = sum;

		}
	__global__ 
		void 
		matrix_mul_kernel(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension)
		{

			int b_col = blockIdx.x;
			int b_row = blockIdx.y;
			int t_col = threadIdx.x;
			int t_row = threadIdx.y;

			float *sub_matrix = get_sub_matrix(sq_matrix_result, sq_dimension, b_row, b_col);

			float sum = 0.0f;

			for(int i = 0; i < sq_dimension/BLOCK_SIZE; i++)
			{
				float *sub_1 = get_sub_matrix(sq_matrix_1, sq_dimension, b_row, i);
				float *sub_2 = get_sub_matrix(sq_matrix_2, sq_dimension, i, b_col);
				__shared__ float A[BLOCK_SIZE][BLOCK_SIZE];
				__shared__ float B[BLOCK_SIZE][BLOCK_SIZE];

				A[t_row][t_col] = sub_1[t_row*sq_dimension+t_col];
				B[t_row][t_col] = sub_2[t_row*sq_dimension+t_col]; 

				__syncthreads();

				for (int j = 0; j < BLOCK_SIZE ; ++j){
					sum += A[t_row][j] * B[j][t_col]; 
				}

				__syncthreads();


			}
			sub_matrix[t_row*sq_dimension + t_col] = sum;

		}

	void 
		matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
		{

			/* padded dimension*/
			unsigned int new_dimension = (sq_dimension%BLOCK_SIZE == 0? sq_dimension:(sq_dimension/BLOCK_SIZE+1)*BLOCK_SIZE);
			unsigned int size = new_dimension * new_dimension * sizeof(float);
			float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;
			/***************************************************
			  0st Part: padding, if necessary
			 ****************************************************/
			float *padded_matrix_1, *padded_matrix_2, *padded_matrix_result;
			if(sq_dimension ^ new_dimension){
				padded_matrix_1 = (float *)malloc(size);
				padded_matrix_2 = (float *)malloc(size);
				padded_matrix_result = (float *)malloc(size);

				padding(sq_matrix_1, padded_matrix_1, sq_dimension, new_dimension);
				padding(sq_matrix_2, padded_matrix_2,  sq_dimension, new_dimension);
			}else{
				padded_matrix_1 = sq_matrix_1;
				padded_matrix_2 = sq_matrix_2;
				padded_matrix_result = sq_matrix_result;
			}
			/***************************************************
			  1st Part: Allocation of memory on device memory  
			 ****************************************************/

			/* copy sq_matrix_1 and sq_matrix_2 to device memory */
			hipMalloc((void**) &sq_matrix_1_d, size);
			hipMemcpy(sq_matrix_1_d, padded_matrix_1, size, hipMemcpyHostToDevice);
			hipMalloc((void**) &sq_matrix_2_d, size);
			hipMemcpy(sq_matrix_2_d, padded_matrix_2, size, hipMemcpyHostToDevice);

			/*allocate sq_matrix_result on host */
			hipMalloc((void**) &sq_matrix_result_d, size);

			/***************************************************
			  2nd Part: Inovke kernel 
			 ****************************************************/
			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
			dim3 dimGrid(new_dimension / dimBlock.x, new_dimension / dimBlock.y);
			if(sq_dimension > BLOCK_SIZE){
				matrix_mul_kernel<<<dimGrid, dimBlock, dimBlock.x * dimBlock.x * sizeof(float)>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, new_dimension);
			}
			else{
				small_matrix_mul_kernel<<<dimGrid, dimBlock, dimBlock.x * dimBlock.x * sizeof(float)>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, new_dimension);
			}
			/***************************************************
			  3rd Part: Transfer result from device to host 
			 ****************************************************/
			hipMemcpy(padded_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);

			/***************************************************
			  4th Part: free the padded matrices, if padding is necessary
			 ****************************************************/
			if(sq_dimension ^ new_dimension){
				padding_reverse(sq_matrix_result, padded_matrix_result, sq_dimension, new_dimension);
				free(padded_matrix_1);
				free(padded_matrix_2);
				free(padded_matrix_result);
			}
			hipFree(sq_matrix_1_d);
			hipFree(sq_matrix_2_d);
			hipFree(sq_matrix_result_d);
		}  
} // namespace cuda
